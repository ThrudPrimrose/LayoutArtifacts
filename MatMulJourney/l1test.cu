#include <hip/hip_runtime.h>
#include <iostream>

constexpr long long N = 1024 * 1024 * 512;
constexpr long long E = 4;

#define checkCudaError(ans)                   \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

template <long long LoadSize, long long NumThreads>
__device__ __forceinline__ void loadToSharedMemory(const float *input, float *sharedMem, const long long globalOffset)
{
    static_assert(LoadSize % NumThreads == 0 && LoadSize >= NumThreads);
    const long long index = threadIdx.x;

    for (long long i = 0; i < LoadSize; i += NumThreads)
    {
        sharedMem[index + i] = input[globalOffset + index + i];
    }

    __syncthreads();
}

template <long long NumThreads>
__global__ void sum4_front(const float *input, float *output, long long vectorSize)
{
    const long long startOffset = blockIdx.x * NumThreads * E;
    const long long globalIdx = blockIdx.x * NumThreads + threadIdx.x;
    const long long startIdx = threadIdx.x * E;

    __shared__ float tile[E * NumThreads];
    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset);

    float sum = 0.0f;
    const float4 *access = reinterpret_cast<const float4 *>(&tile[startIdx]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx] = sum;
}

template <long long NumThreads>
__global__ void sum4_back(const float *input, float *output, long long vectorSize)
{
    const long long startOffset = N - ((1 + blockIdx.x) * NumThreads * E);
    const long long globalIdx = ((N / E) - ((1 + blockIdx.x) * NumThreads)) + threadIdx.x;
    const long long startIdx = threadIdx.x * E;

    __shared__ float tile[E * NumThreads];
    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset);

    float sum = 0.0f;
    const float4 *access = reinterpret_cast<const float4 *>(&tile[startIdx]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx] = sum;
}

template <long long NumThreads>
__global__ void sum4_front_and_back(const float *input, float *output, long long vectorSize)
{
    const long long startOffset = blockIdx.x * NumThreads;
    const long long globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    const long long startIdx = threadIdx.x * E;

    __shared__ float tile[E * NumThreads];
    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset);

    float sum = 0.0f;
    const float4 *access = nullptr;

    access = reinterpret_cast<const float4 *>(&tile[startIdx]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx] = sum;

    __syncthreads();

    const long long startOffset2 = N - ((1 + blockIdx.x) * NumThreads * E);
    const long long globalIdx2 = ((N / E) - ((1 + blockIdx.x) * NumThreads)) + threadIdx.x;
    const long long startIdx2 = threadIdx.x * E;

    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset2);

    access = reinterpret_cast<const float4 *>(&tile[startIdx2]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx2] = sum;
}

template <long long NumThreads>
__global__ void sum4_front_and_front(const float *input, float *output, long longvectorSize)
{
    const long long startOffset = blockIdx.x * NumThreads;
    const long long globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    const long long startIdx = threadIdx.x * E;

    __shared__ float tile[E * NumThreads];
    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset);

    float sum = 0.0f;
    const float4 *access = nullptr;

    access = reinterpret_cast<const float4 *>(&tile[startIdx]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx] = sum;

    __syncthreads();

    const long long startOffset2 = blockIdx.x * NumThreads;
    const long long globalIdx2 = blockIdx.x * blockDim.x + threadIdx.x;
    const long long startIdx2 = threadIdx.x * E;

    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset2);

    access = reinterpret_cast<const float4 *>(&tile[startIdx2]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx2] = sum;
}

template <long long NumThreads>
__global__ void sum4_front_and_front_double(const float *input, const float *input2, float *output, long longvectorSize)
{
    const long long startOffset = blockIdx.x * NumThreads;
    const long long globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    const long long startIdx = threadIdx.x * E;

    __shared__ float tile[E * NumThreads];
    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset);

    float sum = 0.0f;
    const float4 *access = nullptr;

    access = reinterpret_cast<const float4 *>(&tile[startIdx]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx] = sum;

    __syncthreads();



    const long long startOffset2 = blockIdx.x * NumThreads;
    const long long globalIdx2 = blockIdx.x * blockDim.x + threadIdx.x;
    const long long startIdx2 = threadIdx.x * E;
    loadToSharedMemory<E * NumThreads, NumThreads>(input2, tile, startOffset2);

    access = reinterpret_cast<const float4 *>(&tile[startIdx2]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx2] = sum;
}

template <long long NumThreads>
__global__ void sum4_front_and_back_double(const float *input, const float *input2, float *output, long longvectorSize)
{
    const long long startOffset = blockIdx.x * NumThreads;
    const long long globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    const long long startIdx = threadIdx.x * E;

    __shared__ float tile[E * NumThreads];
    loadToSharedMemory<E * NumThreads, NumThreads>(input, tile, startOffset);

    float sum = 0.0f;
    const float4 *access = nullptr;

    access = reinterpret_cast<const float4 *>(&tile[startIdx]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx] = sum;

    __syncthreads();

    const long long startOffset2 = N - ((1 + blockIdx.x) * NumThreads * E);
    const long long globalIdx2 = ((N / E) - ((1 + blockIdx.x) * NumThreads)) + threadIdx.x;
    const long long startIdx2 = threadIdx.x * E;
    loadToSharedMemory<E * NumThreads, NumThreads>(input2, tile, startOffset2);

    access = reinterpret_cast<const float4 *>(&tile[startIdx2]);
    sum = access->x + access->w + access->y + access->z;
    output[globalIdx2] = sum;
}

int main()
{
    // Host vector
    static_assert(N % E == 0);
    float *h_input = new float[N];
    float *h_input2 = new float[N];

    float *d_input, *d_output, *d_input2;

    constexpr long long NT = 256;

    hipEvent_t start, stop;
    float milliseconds = 0;

    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));
    std::cout << "Let's start" << std::endl;

    checkCudaError(hipMalloc(&d_input, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_input2, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_output, (N / E) * sizeof(float)));

    checkCudaError(hipEventRecord(start));
    sum4_front<NT><<<N / (4 * NT), NT>>>(d_input, d_output, N);
    sum4_front<NT><<<N / (4 * NT), NT>>>(d_input, d_output, N);
    checkCudaError(hipDeviceSynchronize());
    checkCudaError(hipEventRecord(stop));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time for sum4_front and sum4_front: " << milliseconds << " ms" << std::endl;
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    hipFree(d_input);
    hipFree(d_input2);
    hipFree(d_output);
    hipDeviceReset();
    checkCudaError(hipMalloc(&d_input, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_input2, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_output, (N / E) * sizeof(float)));
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    checkCudaError(hipEventRecord(start));
    sum4_front<NT><<<N / (4 * NT), NT>>>(d_input, d_output, N);
    sum4_back<NT><<<N / (4 * NT), NT>>>(d_input, d_output, N);
    checkCudaError(hipDeviceSynchronize());
    checkCudaError(hipEventRecord(stop));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time for sum4_front and sum4_back: " << milliseconds << " ms" << std::endl;
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    hipFree(d_input);
    hipFree(d_input2);
    hipFree(d_output);
    hipDeviceReset();
    checkCudaError(hipMalloc(&d_input, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_input2, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_output, (N / E) * sizeof(float)));
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    checkCudaError(hipEventRecord(start));
    sum4_front_and_back<NT><<<N / (4 * NT), NT>>>(d_input, d_output, N);
    checkCudaError(hipDeviceSynchronize());
    checkCudaError(hipEventRecord(stop));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time for sum4_front_and_back: " << milliseconds << " ms" << std::endl;
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    hipFree(d_input);
    hipFree(d_input2);
    hipFree(d_output);
    hipDeviceReset();
    checkCudaError(hipMalloc(&d_input, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_input2, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_output, (N / E) * sizeof(float)));
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    checkCudaError(hipEventRecord(start));
    sum4_front_and_front<NT><<<N / (4 * NT), NT>>>(d_input, d_output, N);
    checkCudaError(hipDeviceSynchronize());
    checkCudaError(hipEventRecord(stop));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time for sum4_front_and_front: " << milliseconds << " ms" << std::endl;
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    hipFree(d_input);
    hipFree(d_input2);
    hipFree(d_output);
    hipDeviceReset();
    checkCudaError(hipMalloc(&d_input, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_input2, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_output, (N / E) * sizeof(float)));
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    checkCudaError(hipEventRecord(start));
    sum4_front_and_front_double<NT><<<N / (4 * NT), NT>>>(d_input, d_input2, d_output, N);
    checkCudaError(hipDeviceSynchronize());
    checkCudaError(hipEventRecord(stop));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time for sum4_front_and_front_double: " << milliseconds << " ms" << std::endl;
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    hipFree(d_input);
    hipFree(d_input2);
    hipFree(d_output);
    hipDeviceReset();
    checkCudaError(hipMalloc(&d_input, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_input2, N * sizeof(float)));
    checkCudaError(hipMalloc(&d_output, (N / E) * sizeof(float)));
    checkCudaError(hipEventCreate(&start));
    checkCudaError(hipEventCreate(&stop));

    checkCudaError(hipEventRecord(start));
    sum4_front_and_back_double<NT><<<N / (4 * NT), NT>>>(d_input, d_input2, d_output, N);
    checkCudaError(hipDeviceSynchronize());
    checkCudaError(hipEventRecord(stop));
    checkCudaError(hipEventSynchronize(stop));
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Time for sum4_front_and_back_double: " << milliseconds << " ms" << std::endl;

    // Cleanup
    checkCudaError(hipEventDestroy(start));
    checkCudaError(hipEventDestroy(stop));

    // Clean up

    delete[] h_input;

    return 0;
}