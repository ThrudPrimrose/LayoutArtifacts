#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <fstream>
#include <sstream>

#include "util.h"
#include "device_properties.cuh"
#include <hipblas.h>
#include "2dblocktiling.cuh"
#include "2dblocktiling2.cuh"
#include "2dblocktiling3.cuh"
#include "2dblocktiling4.cuh"
#include "2dblocktiling5.cuh"
#include "2dblocktiling6.cuh"
#include "2dblocktiling7.cuh"
#include "cg.cuh"

#ifdef PTY
constexpr uint tY = PTY;
#else
constexpr uint tY = 16;
#endif
#ifdef PTY
constexpr uint tX = PTX;
#else
constexpr uint tX = 16;
#endif
#ifdef PBK
constexpr uint bK = PBK;
#else
constexpr uint bK = 16;
#endif
#ifdef PTM
constexpr uint tM = PTM;
#else
constexpr uint tM = 8;
#endif
#ifdef PTN
constexpr uint tN = PTN;
#else
constexpr uint tN = 8;
#endif
static_assert(tN != 0);
static_assert(tM != 0);
static_assert(tX != 0);
static_assert(tY != 0);
static_assert(bK != 0);

// Define matrix dimensions as constexpr
constexpr uint64_t M = 16384;
constexpr uint64_t K = 16384;
constexpr uint64_t N = 16384;
constexpr double ops = M * N * K * 2;
constexpr double byte_accessed = sizeof(float) * (M * N + K * N + M * K);

#define RUN_AND_MEASURE(name, file, kernel)                                                                                         \
    {                                                                                                                               \
        hipEvent_t beg, end;                                                                                                       \
        CHECK_CUDA_ERR(hipEventCreate(&beg));                                                                                      \
        CHECK_CUDA_ERR(hipEventCreate(&end));                                                                                      \
        float ms;                                                                                                                   \
        CHECK_CUDA_ERR(hipEventRecord(beg));                                                                                       \
        kernel();                                                                                                                   \
        CHECK_CUDA_ERR(hipEventRecord(end));                                                                                       \
        CHECK_CUDA_ERR(hipDeviceSynchronize());                                                                                    \
        CHECK_CUDA_ERR(hipEventElapsedTime(&ms, beg, end));                                                                        \
        hipError_t err = hipGetLastError();                                                                                       \
        if (err != hipSuccess)                                                                                                     \
        {                                                                                                                           \
            printf("Kernel launch error: %s\n", hipGetErrorString(err));                                                           \
        }                                                                                                                           \
        hipDeviceSynchronize();                                                                                                    \
        CHECK_CUDA_ERR(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));                                                       \
        bool print = true;                                                                                                          \
        float max_diff = 0.f;                                                                                                       \
        for (size_t i = 0; i < M * N; i++)                                                                                          \
        {                                                                                                                           \
            float diff = ref_C[i] - h_C[i];                                                                                         \
            float abs_diff = diff > 0.f ? diff : -diff;                                                                             \
            if (abs_diff > max_diff)                                                                                                \
            {                                                                                                                       \
                max_diff = abs_diff;                                                                                                \
            }                                                                                                                       \
            if (print && (diff < -5.0 || diff > 5.0))                                                                               \
            {                                                                                                                       \
                std::cout << ("Difference too huge at " + std::to_string(i) +                                                       \
                              " for (" + name + ") value should be: " + std::to_string(ref_C[i]) + " is " + std::to_string(h_C[i])) \
                          << std::endl;                                                                                             \
                print = false;                                                                                                      \
            }                                                                                                                       \
        }                                                                                                                           \
        {                                                                                                                           \
            std::cout << "Max diff for (" << name << "): " << std::to_string(max_diff) << std::endl;                                \
            print = false;                                                                                                          \
        }                                                                                                                           \
                                                                                                                                    \
        double time = static_cast<double>(ms) * 1e-3;                                                                               \
        constexpr double op_intensity = ops / byte_accessed;                                                                        \
        double bandwidth, flops;                                                                                                    \
        get_max_bandwidth_and_flops(bandwidth, flops);                                                                              \
        double best_perf = std::min(op_intensity * bandwidth, flops);                                                               \
                                                                                                                                    \
        std::cout << "Peak Achievable perf.:" << best_perf << "GFLOP/s" << std::endl;                                               \
        double achieved_perf = (ops * 1e-9) / time;                                                                                 \
        std::cout << "Achieved perf. of kernel " << name << ": " << achieved_perf << "GFLOP/s" << std::endl;                        \
        double percentage_of_peak = (100.0 * achieved_perf) / best_perf;                                                            \
        std::cout << percentage_of_peak << "% of the achievable peak" << std::endl;                                                 \
        CHECK_CUDA_ERR(hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice));                                                       \
        std::cout << std::endl                                                                                                      \
                  << std::endl;                                                                                                     \
        file << name << ','                                                                                                         \
             << std::to_string(tY) << ','                                                                                           \
             << std::to_string(tX) << ','                                                                                           \
             << std::to_string(bK) << ','                                                                                           \
             << std::to_string(tM) << ','                                                                                           \
             << std::to_string(tN) << ','                                                                                           \
             << ((err != hipSuccess) ? "-1.0" : std::to_string(time)) << ','                                                       \
             << ((err != hipSuccess) ? "-1.0" : std::to_string(achieved_perf)) << ','                                              \
             << ((err != hipSuccess) ? "-1.0" : std::to_string(percentage_of_peak)) << '\n';                                       \
    }

int main()
{
    // Allocate host memory
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    float *h_A = (float *)malloc(size_A);
    float *h_B = (float *)malloc(size_B);
    float *h_C = (float *)malloc(size_C);
    float *ref_C = (float *)malloc(size_C);

    // Read matrices A and B from binary files
    readBinaryFile("matrix_A.bin", h_A, size_A);
    readBinaryFile("matrix_B.bin", h_B, size_B);
    readBinaryFile("matrix_C_ref.bin", ref_C, size_C);

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    CHECK_CUDA_ERR(hipMalloc((void **)&d_A, size_A));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_B, size_B));
    CHECK_CUDA_ERR(hipMalloc((void **)&d_C, size_C));

    // Copy matrices A and B to device
    CHECK_CUDA_ERR(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    // auto naive = [d_A, d_B, d_C, ref_C](){ matmul_naive_launcher<32, 32>(d_A, d_B, d_C, M, N, K); hipMemcpy(ref_C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost); };
    // RUN_AND_MEASURE("naive", naive);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    auto cublas = [d_A, d_B, d_C, ref_C, &handle]()
    {
        float alpha = 1.0f;
        float beta = 0.0f;
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    M, N, K,
                    &alpha,
                    d_B, K,
                    d_A, M,
                    &beta,
                    d_C, M);
        CHECK_CUDA_ERR(hipMemcpy(ref_C, d_C, M * N, hipMemcpyDeviceToHost));
    };
    std::string identifier_cublas("cuBLAS");
    std::string identifier_1("Outer-Product + No Remainder");
    std::string identifier_2("Inner Product + No Remainder");
    std::string identifier_3("Inner Product + Remainder");
    std::string identifier_4("Inner Product DaCe style");
    std::string identifier_5("Inner Product + Store");
    std::string identifier_6("Inner Product + Store (LL1)");
    std::string identifier_7("Inner Product + Store (LL2)");
    std::string identifier_8("Inner Product + Store (LL4)");
    std::string identifier_9("Inner Product + Store (LL8)");
    std::string identifier_10("Inner Product + Store (LL16)");
    std::string identifier_11("Inner Product + Store + Mem Reuse (LL1)");
    std::string identifier_12("Inner Product + Store + Mem Reuse (LL2)");
    std::string identifier_13("Inner Product + Store + Mem Reuse (LL4)");
    std::string identifier_14("Inner Product + Store + Mem Reuse (LL8)");
    std::string identifier_15("Inner Product + Store + Mem Reuse (LL16)");
    std::string identifier_cg("Inner Product + MiG?");
    std::ofstream _file("output.csv", std::ios::app);
    int sharedMemoryPerBlock;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    sharedMemoryPerBlock = deviceProp.sharedMemPerBlock;
    std::cout << "Device " << 0 << " has " << sharedMemoryPerBlock
              << " bytes of shared memory per block." << std::endl;
    constexpr int bm = tY * tM;
    constexpr int bn = tX * tN;
    std::cout << "Too much shared memory used? " << (4 * bm * bK + 4 * bK * bn + 4 * bm * bn > sharedMemoryPerBlock) << std::endl
              << std::endl;
    RUN_AND_MEASURE(identifier_cublas, _file, cublas);
    // auto __2d_block_tiling = [d_A, d_B, d_C](){ _2d_block_tiling_launcher<tY, tX, bK, tM, tN>(d_A, d_B, d_C, M, N, K); };
    // RUN_AND_MEASURE(identifier_1, _file, __2d_block_tiling);
    auto __2d_block_tiling2 = [d_A, d_B, d_C]()
    { _2d_block_tiling_launcher2<tY, tX, bK, tM, tN>(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_2, _file, __2d_block_tiling2);
    // auto __2d_block_tiling3 = [d_A, d_B, d_C](){ _2d_block_tiling_launcher3<tY, tX, bK, tM, tN>(d_A, d_B, d_C, M, N, K); };
    // RUN_AND_MEASURE(identifier_3, _file, __2d_block_tiling3);
    auto __2d_block_tiling4 = [d_A, d_B, d_C]()
    { __dace_runkernel_dace_naive_matmul_258_0_0_3(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_4, _file, __2d_block_tiling4);
    //auto __2d_block_tiling5 = [d_A, d_B, d_C]()
    //{ _2d_block_tiling_launcher5<tY, tX, bK, tM, tN / 2>(d_A, d_B, d_C, M, N, K); };
    //RUN_AND_MEASURE(identifier_5, _file, __2d_block_tiling5);
    auto __2d_block_tiling6 = [d_A, d_B, d_C]()
    { _2d_block_tiling_launcher6<tY, tX, bK, tM, tN, 1>(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_6, _file, __2d_block_tiling6);
    auto __2d_block_tiling7 = [d_A, d_B, d_C]()
    { _2d_block_tiling_launcher6<tY, tX, bK, tM, tN, 2>(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_7, _file, __2d_block_tiling7);
    auto __2d_block_tiling8 = [d_A, d_B, d_C]()
    { _2d_block_tiling_launcher6<tY, tX, bK, tM, tN, 4>(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_8, _file, __2d_block_tiling8);
    // auto __2d_block_tiling9 = [d_A, d_B, d_C](){  _2d_block_tiling_launcher6<tY, tX, bK, tM, tN/2, 8>(d_A, d_B, d_C, M, N, K); };
    // RUN_AND_MEASURE(identifier_9, _file, __2d_block_tiling9);

    auto __2d_block_tiling11 = [d_A, d_B, d_C]()
    { _2d_block_tiling_launcher7<tY, tX, bK, tM, tN, 1>(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_11, _file, __2d_block_tiling11);
    auto __2d_block_tiling12 = [d_A, d_B, d_C]()
    { _2d_block_tiling_launcher7<tY, tX, bK, tM, tN, 2>(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_12, _file, __2d_block_tiling12);
    auto __2d_block_tiling13 = [d_A, d_B, d_C]()
    { _2d_block_tiling_launcher7<tY, tX, bK, tM, tN, 4>(d_A, d_B, d_C, M, N, K); };
    RUN_AND_MEASURE(identifier_13, _file, __2d_block_tiling13);
    // auto __2d_block_tiling14 = [d_A, d_B, d_C](){  _2d_block_tiling_launcher7<tY, tX, bK, tM, tN, 8>(d_A, d_B, d_C, M, N, K); };
    // RUN_AND_MEASURE(identifier_14, _file, __2d_block_tiling14);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    auto cg_lambda = [d_A, d_B, d_C, &stream1, &stream2](){
        matmul_cg_launcher<tY, tX, bK, tM, tN>(stream1, d_A, d_B + (N/2), d_C + (N/2), M, N/2, K, K, N, N);
        matmul_cg_launcher<tY, tX, bK, tM, tN>(stream2, d_A, d_B, d_C, M, N/2, K, K, N, N);
    };
    RUN_AND_MEASURE(identifier_cg, _file, cg_lambda);

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    CHECK_CUDA_ERR(hipFree(d_A));
    CHECK_CUDA_ERR(hipFree(d_B));
    CHECK_CUDA_ERR(hipFree(d_C));
    return 0;
}

// ault[23-24]	1.5 TB		32/64	Intel(R) 6130 @ 2.10GHz		4	NV V100(16GB PCIe)
//
// intelv100 up    1-infini    4:00:00     64 2:16:2       1 idle       ault24