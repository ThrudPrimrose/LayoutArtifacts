#include "hip/hip_runtime.h"
#pragma once
#include "cuda_util.cuh"
#include <mma.h>
using namespace nvcuda;

template <const int WarpsY, 
          const int WarpsX,
          const int BK,
          const int WM, 
          const int WN>
__global__ void _matmul_tensor_core(const half* __restrict__ A, const half* __restrict__ B, half* __restrict__ C,
                                       const int M, const int N, const int K) {
  constexpr int numWarps = WarpsY * WarpsX;
  const int threadId = threadIdx.x;
  const int warpId = threadId / 32;
  const int warpX = warpId % WarpsX;
  const int warpY = warpId % WarpsY;
  const int m = blockIdx.y * WarpsY * WM * 16 + warpY * WM * 16;
  const int n = blockIdx.x * WarpsX * WN * 16 + warpX * WN * 16;


  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag[WM * WN];
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag[WM * WN];
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> acc_frag[WM * WN];
  wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag[WM * WN];

    #pragma unroll
    for (int wi = 0; wi < WM; wi++){
        #pragma unroll
        for (int wj = 0; wj < WN; wj++){
            wmma::fill_fragment(acc_frag[wi * WN + wj], 0.0f);
        }
    }


  for (int k = 0; k < K; k+=16){
    #pragma unroll
    for (int wi = 0; wi < WM; wi++){
        #pragma unroll
        for (int wj = 0; wj < WN; wj++){
          int aRow = m + wi * 16;
          int aCol = k;
          int bRow = k;
          int bCol = n + wj * 16;

          wmma::load_matrix_sync(a_frag, A + aRow + aCol * N, N);
          wmma::load_matrix_sync(b_frag, B + bRow + bCol * K, K);

          wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }
  }


}

template <int WarpsY, int WarpsX, int BK, int WM, int WN>
void matmul_tensor_core_launcher(const half* __restrict__ A,
                                 const half* __restrict__ B, 
                                 half* __restrict__ C,
                                 const int M, const int N, const int K) {
    constexpr int BN = WarpsX * WN * 16;
    constexpr int BM = WarpsY * WM * 16;
    dim3 grid((N + BN - 1) / BN, (M + BM - 1) / BM);
    dim3 block(WarpsX*WarpsY*32, 1);
    _matmul_tensor_core<WarpsY, WarpsX, BK, WM, WN><<<grid, block>>>(A, B, C, M, N, K);
}