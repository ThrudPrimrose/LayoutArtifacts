#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <dace/dace.h>


struct complex_gemm_soa_merged_auto_tiled_0_c2_state_t {
    dace::cuda::Context *gpu_context;
    dace::perf::Report report;
};


#include <chrono>
#include <hip/hip_runtime.h>


DACE_EXPORTED int __dace_init_cuda(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state, int K, int M, int N);
DACE_EXPORTED int __dace_exit_cuda(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state);



DACE_EXPORTED int __dace_init_cuda(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state, int K, int M, int N) {
    int count;

    // Check that we are able to run cuda code
    if (hipGetDeviceCount(&count) != hipSuccess)
    {
        printf("ERROR: GPU drivers are not configured or cuda-capable device "
               "not found\n");
        return 1;
    }
    if (count == 0)
    {
        printf("ERROR: No cuda-capable devices found\n");
        return 2;
    }

    // Initialize cuda before we run the application
    float *dev_X;
    DACE_GPU_CHECK(hipMalloc((void **) &dev_X, 1));
    DACE_GPU_CHECK(hipFree(dev_X));



    __state->gpu_context = new dace::cuda::Context(4, 11);

    // Create cuda streams and events
    for(int i = 0; i < 4; ++i) {
        DACE_GPU_CHECK(hipStreamCreateWithFlags(&__state->gpu_context->internal_streams[i], hipStreamNonBlocking));
        __state->gpu_context->streams[i] = __state->gpu_context->internal_streams[i]; // Allow for externals to modify streams
    }
    for(int i = 0; i < 11; ++i) {
        DACE_GPU_CHECK(hipEventCreateWithFlags(&__state->gpu_context->events[i], hipEventDisableTiming));
    }



    return 0;
}

DACE_EXPORTED int __dace_exit_cuda(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state) {


    // Synchronize and check for CUDA errors
    int __err = static_cast<int>(__state->gpu_context->lasterror);
    if (__err == 0)
        __err = static_cast<int>(hipDeviceSynchronize());

    // Destroy cuda streams and events
    for(int i = 0; i < 4; ++i) {
        DACE_GPU_CHECK(hipStreamDestroy(__state->gpu_context->internal_streams[i]));
    }
    for(int i = 0; i < 11; ++i) {
        DACE_GPU_CHECK(hipEventDestroy(__state->gpu_context->events[i]));
    }

    delete __state->gpu_context;
    return __err;
}

DACE_EXPORTED bool __dace_gpu_set_stream(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state, int streamid, gpuStream_t stream)
{
    if (streamid < 0 || streamid >= 4)
        return false;

    __state->gpu_context->streams[streamid] = stream;

    return true;
}

DACE_EXPORTED void __dace_gpu_set_all_streams(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state, gpuStream_t stream)
{
    for (int i = 0; i < 4; ++i)
        __state->gpu_context->streams[i] = stream;
}

__global__ void __launch_bounds__(256) GPU_DeviceMap_0_0_30(const float * __restrict__ Aim, const float * __restrict__ Ar, const float * __restrict__ Bim, const float * __restrict__ Br, float * __restrict__ Cim, float * __restrict__ Cr, int K, int M, int N) {
    {
        {
            int b_j = (64 * blockIdx.x);
            int b_i = (16 * blockIdx.y);
            {
                {
                    {
                        int d_j = ((2 * threadIdx.x) + b_j);
                        int d_i = ((2 * threadIdx.y) + b_i);
                        if (d_j >= b_j) {
                            if (d_i >= b_i) {
                                {
                                    for (auto k = 0; k < K; k += 16) {
                                        __shared__ float shr_Bim[1024];
                                        __shared__ float shr_Aim[256];
                                        __shared__ float shr_Br[1024];
                                        __shared__ float shr_Ar[256];
                                        {
                                            const float* IN_Bim = &Bim[((N * k) + d_j)];
                                            float* OUT_Bim = shr_Bim;

                                            ///////////////////
                                            // Bim[K,N]
                                            // shr_Bim[16,64]
                                            // Strides // shr_Bim[64,1]
                                            // Inner Loop Condition: k <= K - 16 && b_j <= N - 64
                                            const int tid = threadIdx.x + blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
                                            // Num Threads: 256, Line Length (max): 64
                                            // load multiple lines at a time 4
                                            const int line_offset = tid % 64;
                                            const int line_num = tid / 64;
                                            #pragma unroll
                                            for (int i0 = 0; i0 < 16; i0 += 4) {
                                                //64, (64, 1),  + ((i0) * 64)
                                                //N, (N, 1),  + ((i0) * N)
                                                shr_Bim[line_num*64 + line_offset + ((i0) * 64)] = Bim[(N*(k))+(1*(b_j)) + line_num*N + line_offset + ((i0) * N)];
                                            }
                                            ///////////////////

                                        }
                                        {
                                            const float* IN_Aim = &Aim[((K * d_i) + k)];
                                            float* OUT_Aim = shr_Aim;

                                            ///////////////////
                                            // Aim[M,K]
                                            // shr_Aim[16,16]
                                            // Strides // shr_Aim[16,1]
                                            // Inner Loop Condition: b_i <= M - 16 && k <= K - 16
                                            const int tid = threadIdx.x + blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
                                            // Num Threads: 256, Line Length (max): 16
                                            // load multiple lines at a time 16
                                            const int line_offset = tid % 16;
                                            const int line_num = tid / 16;
                                            int i0 = 0;
                                            //16, (16, 1),
                                            //K, (K, 1),
                                            shr_Aim[line_num*16 + line_offset] = Aim[(K*(b_i))+(1*(k)) + line_num*K + line_offset];
                                            ///////////////////

                                        }
                                        {
                                            const float* IN_Br = &Br[((N * k) + d_j)];
                                            float* OUT_Br = shr_Br;

                                            ///////////////////
                                            // Br[K,N]
                                            // shr_Br[16,64]
                                            // Strides // shr_Br[64,1]
                                            // Inner Loop Condition: k <= K - 16 && b_j <= N - 64
                                            const int tid = threadIdx.x + blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
                                            // Num Threads: 256, Line Length (max): 64
                                            // load multiple lines at a time 4
                                            const int line_offset = tid % 64;
                                            const int line_num = tid / 64;
                                            #pragma unroll
                                            for (int i0 = 0; i0 < 16; i0 += 4) {
                                                //64, (64, 1),  + ((i0) * 64)
                                                //N, (N, 1),  + ((i0) * N)
                                                shr_Br[line_num*64 + line_offset + ((i0) * 64)] = Br[(N*(k))+(1*(b_j)) + line_num*N + line_offset + ((i0) * N)];
                                            }
                                            ///////////////////

                                        }
                                        {
                                            const float* IN_Ar = &Ar[((K * d_i) + k)];
                                            float* OUT_Ar = shr_Ar;

                                            ///////////////////
                                            // Ar[M,K]
                                            // shr_Ar[16,16]
                                            // Strides // shr_Ar[16,1]
                                            // Inner Loop Condition: b_i <= M - 16 && k <= K - 16
                                            const int tid = threadIdx.x + blockDim.x * threadIdx.y + (blockDim.x * blockDim.y) * threadIdx.z;
                                            // Num Threads: 256, Line Length (max): 16
                                            // load multiple lines at a time 16
                                            const int line_offset = tid % 16;
                                            const int line_num = tid / 16;
                                            int i0 = 0;
                                            //16, (16, 1),
                                            //K, (K, 1),
                                            shr_Ar[line_num*16 + line_offset] = Ar[(K*(b_i))+(1*(k)) + line_num*K + line_offset];
                                            __syncthreads();
                                            ///////////////////

                                        }
                                        {
                                            #pragma unroll
                                            for (auto i = d_i; i < (d_i + 2); i += 1) {
                                                #pragma unroll
                                                for (auto j = d_j; j < (d_j + 2); j += 1) {
                                                    {
                                                        #pragma unroll
                                                        for (auto tk = 0; tk < 16; tk += 1) {
                                                            float __tmp2;
                                                            float tmp1;
                                                            float __tmp4;
                                                            float tmp2;
                                                            float __tmp6;
                                                            float tmp3 = 0;
                                                            float __tmp8;
                                                            float tmp4 = 0;
                                                            {
                                                                float __in2 = shr_Bim[(((- d_j) + j) + (64 * tk))];
                                                                float __in1 = shr_Aim[(((-16 * d_i) + (16 * i)) + tk)];
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Mult_)
                                                                __out = (__in1 * __in2);
                                                                ///////////////////

                                                                __tmp4 = __out;
                                                            }
                                                            {
                                                                float __in2 = __tmp4;
                                                                float __in1 = Cim[((N * i) + j)];
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Sub_)
                                                                __out = (__in1 - __in2);
                                                                ///////////////////

                                                                tmp2 = __out;
                                                            }
                                                            {
                                                                float __in2 = shr_Bim[(((- d_j) + j) + (64 * tk))];
                                                                float __in1 = shr_Ar[(((-16 * d_i) + (16 * i)) + tk)];
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Mult_)
                                                                __out = (__in1 * __in2);
                                                                ///////////////////

                                                                __tmp8 = __out;
                                                            }
                                                            {
                                                                float __in1 = tmp2;
                                                                float __in2 = __tmp8;
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Add_)
                                                                __out = (__in1 + __in2);
                                                                ///////////////////

                                                                tmp4 = __out;
                                                            }
                                                            {
                                                                float _in = tmp4;
                                                                float _out;

                                                                ///////////////////
                                                                // Tasklet code (assign)
                                                                _out = _in;
                                                                ///////////////////

                                                                Cim[((N * i) + j)] = _out;
                                                            }
                                                            {
                                                                float __in1 = shr_Aim[(((-16 * d_i) + (16 * i)) + tk)];
                                                                float __in2 = shr_Br[(((- d_j) + j) + (64 * tk))];
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Mult_)
                                                                __out = (__in1 * __in2);
                                                                ///////////////////

                                                                __tmp6 = __out;
                                                            }
                                                            {
                                                                float __in2 = shr_Br[(((- d_j) + j) + (64 * tk))];
                                                                float __in1 = shr_Ar[(((-16 * d_i) + (16 * i)) + tk)];
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Mult_)
                                                                __out = (__in1 * __in2);
                                                                ///////////////////

                                                                __tmp2 = __out;
                                                            }
                                                            {
                                                                float __in2 = __tmp2;
                                                                float __in1 = Cr[((N * i) + j)];
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Add_)
                                                                __out = (__in1 + __in2);
                                                                ///////////////////

                                                                tmp1 = __out;
                                                            }
                                                            {
                                                                float __in1 = tmp1;
                                                                float __in2 = __tmp6;
                                                                float __out;

                                                                ///////////////////
                                                                // Tasklet code (_Add_)
                                                                __out = (__in1 + __in2);
                                                                ///////////////////

                                                                tmp3 = __out;
                                                            }
                                                            {
                                                                float _in = tmp3;
                                                                float _out;

                                                                ///////////////////
                                                                // Tasklet code (assign)
                                                                _out = _in;
                                                                ///////////////////

                                                                Cr[((N * i) + j)] = _out;
                                                            }
                                                        }
                                                    }
                                                }
                                            }
                                        }
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}


DACE_EXPORTED void __dace_runkernel_GPU_DeviceMap_0_0_30(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state, const float * __restrict__ Aim, const float * __restrict__ Ar, const float * __restrict__ Bim, const float * __restrict__ Br, float * __restrict__ Cim, float * __restrict__ Cr, int K, int M, int N);
void __dace_runkernel_GPU_DeviceMap_0_0_30(complex_gemm_soa_merged_auto_tiled_0_c2_state_t *__state, const float * __restrict__ Aim, const float * __restrict__ Ar, const float * __restrict__ Bim, const float * __restrict__ Br, float * __restrict__ Cim, float * __restrict__ Cr, int K, int M, int N)
{

    if ((int_ceil(N, 64)) == 0 || (int_ceil(M, 16)) == 0) {

        return;
    }

    void  *GPU_DeviceMap_0_0_30_args[] = { (void *)&Aim, (void *)&Ar, (void *)&Bim, (void *)&Br, (void *)&Cim, (void *)&Cr, (void *)&K, (void *)&M, (void *)&N };
    gpuError_t __err = hipLaunchKernel((void*)GPU_DeviceMap_0_0_30, dim3(int_ceil(N, 64), int_ceil(M, 16), 1), dim3(32, 8, 1), GPU_DeviceMap_0_0_30_args, 0, __state->gpu_context->streams[0]);
    DACE_KERNEL_LAUNCH_CHECK(__err, "GPU_DeviceMap_0_0_30", int_ceil(N, 64), int_ceil(M, 16), 1, 32, 8, 1);
}

